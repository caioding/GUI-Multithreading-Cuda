#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>

// Kernel CUDA para o filtro de desfoque
__global__ void blurKernel(const unsigned char* in, unsigned char* out, int width, int height, int channels, int radius) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        float r_acc = 0.0f, g_acc = 0.0f, b_acc = 0.0f;
        int pixel_count = 0;

        for (int y = -radius; y <= radius; ++y) {
            for (int x = -radius; x <= radius; ++x) {
                int current_row = row + y;
                int current_col = col + x;

                if (current_row >= 0 && current_row < height && current_col >= 0 && current_col < width) {
                    int idx = (current_row * width + current_col) * channels;
                    r_acc += in[idx + 0];
                    g_acc += in[idx + 1];
                    b_acc += in[idx + 2];
                    pixel_count++;
                }
            }
        }

        int out_idx = (row * width + col) * channels;
        out[out_idx + 0] = static_cast<unsigned char>(r_acc / pixel_count);
        out[out_idx + 1] = static_cast<unsigned char>(g_acc / pixel_count);
        out[out_idx + 2] = static_cast<unsigned char>(b_acc / pixel_count);

        if (channels == 4) {
            out[out_idx + 3] = in[out_idx + 3];
        }
    }
}

// Fun��o wrapper C-style para ser chamada a partir do C++/CLI
extern "C" void apply_cuda_blur(unsigned char* h_img_in, unsigned char* h_img_out, int width, int height, int channels, int blur_radius) {
    size_t img_size = width * height * channels * sizeof(unsigned char);
    unsigned char* d_img_in = nullptr, * d_img_out = nullptr;

    hipMalloc(&d_img_in, img_size);
    hipMalloc(&d_img_out, img_size);

    hipMemcpy(d_img_in, h_img_in, img_size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // Medi��o do tempo do kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    blurKernel << <gridDim, blockDim >> > (d_img_in, d_img_out, width, height, channels, blur_radius);
    hipEventRecord(stop);

    hipDeviceSynchronize();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tempo do kernel CUDA: %.3f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_img_out, d_img_out, img_size, hipMemcpyDeviceToHost);

    hipFree(d_img_in);
    hipFree(d_img_out);
}